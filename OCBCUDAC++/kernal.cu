#include "hip/hip_runtime.h"
#include <stdio.h>
__device__ void test(float * a){
    a[0]=20;
}

__global__ void kernel(float * a, float * b, const int N) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    size_t j = blockIdx.y * blockDim.y + threadIdx.y;
    size_t k = blockIdx.z * blockDim.z + threadIdx.z;
    printf("%f, lleguer \n",a[0]);
    test(a);
    a[0]=20;
}


extern "C" void perform_stencil(float * a, float * b, const int N) {
    hipSetDevice(0);

    float * d_a;
    float * d_b;

    hipEvent_t start, stop;
    float       elapsedTime;
    
    /* begin timing */
    hipEventCreate(&start);
    hipEventRecord(start, 0);

    hipMalloc(&d_a, sizeof(float) * N * N * N);
    hipMalloc(&d_b, sizeof(float) * N * N * N);

    hipMemcpy(d_a, a, sizeof(float) * N * N * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N * N * N, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(1);
    dim3 numBlocks(1);

    // printf("%f\n", d_a[0]);

    kernel <<<numBlocks, threadsPerBlock>>>(d_a, d_b, N);
    hipMemcpy(a,d_a, sizeof(float) * N * N * N, hipMemcpyDeviceToHost);
    /* end timing */
    hipEventCreate(&stop);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);


    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Execution time: %f seconds\n", elapsedTime / 1000);
    hipFree(d_a);
    hipFree(d_b);
}

extern "C" void getDevices() {
    int nDevices;

    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
            prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
            prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
            2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }

}