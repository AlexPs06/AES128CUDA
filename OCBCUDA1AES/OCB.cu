
#include <hip/hip_runtime.h>
#include<iostream>
#include <cstring>
using namespace std;


class aesBlock
{
public:
    unsigned int block[4]={0};
};

unsigned char matrizCajaS[256]={
    0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,
    0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,
    0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,
    0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,
    0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,
    0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,
    0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,
    0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,
    0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,
    0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,
    0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,
    0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,
    0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
    0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,
    0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,
    0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16
};


// 0x8b, 0x2e, 0x18, 0xdd,   
// 0x80, 0x51, 0xfe, 0x46, 
// 0xbc, 0xa9, 0x59, 0x01,
// 0x24, 0xf6, 0xd8, 0x50,
void imprimiArreglo(int tam, unsigned int *in ){

    for (int i = 0; i<tam; i++){
        printf("%08x", in[i] );
    }
}
void ExpansionKeys128( unsigned int *k,unsigned long long klen,  unsigned int keys[11][4] ){
    unsigned char RotWordTemp[4];
    const unsigned int matrizRcon[10]={ 0x01000000, 0x02000000, 0x04000000, 0x08000000, 0x10000000, 0x20000000, 0x40000000, 0x80000000, 0x1b000000, 0x36000000};
    memcpy(&keys[0], k, 16);
    for(int i = 0; i<10; i++){
        RotWordTemp[0]=keys[i][3]>>16;
        RotWordTemp[1]=keys[i][3]>>8;
        RotWordTemp[2]=keys[i][3];
        RotWordTemp[3]=keys[i][3]>>24; 
        
        for(int j = 0;  j < 4; j++ ){
            RotWordTemp[j] = matrizCajaS[ (int) RotWordTemp[j] ];
        }
        int RotWord = 0;
        RotWord = RotWord ^ ( (int) RotWordTemp[0])<<24;
        RotWord = RotWord ^ ( (int) RotWordTemp[1])<<16;
        RotWord = RotWord ^ ( (int) RotWordTemp[2])<<8;
        RotWord = RotWord ^ ( (int) RotWordTemp[3]);
        
        keys[i+1][0] =  RotWord ^ keys[i][0];
        keys[i+1][0] = keys[i+1][0] ^ matrizRcon[i];
        for(int x = 1;  x < 4; x++ ){
            keys[i+1][x] =  keys[i+1][x-1] ^ keys[i][x];
        }
    }
}



__device__ void imprimiArregloCuda(int tam,unsigned char *keys ){
    for (int i = 0; i<tam; i++){
        if(i%4==0)
            printf("\n");
        printf("%x ", keys[i] & 0xff);
    }
}
__device__ void imprimiArregloCudaInt(int tam,unsigned int *keys ){
    printf("----------------\n" );
    for (int i = 0; i<tam; i++){
        printf("%x \n", keys[i] );
    }
}
__device__ void XOR_128(unsigned int* A, unsigned int * B )
{
	for (int i = 0; i < 4; i++) { 
        A[i] =A[i] ^  B[i];
    }
}

__device__ void addRoundKey(unsigned int* in, unsigned int * keys, int round)
{

	for (int i = 0; i < 4; i++) { 
        in[i] =in[i] ^  keys[(round*4) + i];
    }

}


__device__ void subBytes(unsigned int* in, unsigned char * matrizCajaS)
{
    unsigned char * temp;
    temp = (unsigned char *) in;
	for (int i = 0; i < 16; i++) { 
        temp[i] = matrizCajaS[ (int) temp[i]]; 
    }
}



__device__ void shiftRows(unsigned int* in, int *shifttab){
    
    unsigned char * temp;
    unsigned char h[16];
    temp = (unsigned char *) in;
    memcpy(h, temp, 16);

    for(int i = 0; i < 16; i++){
        temp[i] = h[shifttab[i]];
    }
}
__device__ unsigned char GF2Redution(unsigned short in ){
    
    unsigned short temp = in ; // numero de prueba
    // temp = temp * 2;
    unsigned char temp1 = temp>>8; //queda FF
    unsigned char temp2 = temp1<<1; //queda FE
    unsigned char temp3 = temp1<<3; //queda F8
    unsigned char temp4 = temp1<<4; //queda F

    unsigned char temp5 = temp>>13;//queda 7
    unsigned char temp6 = temp5<<1;//queda E 
    unsigned char temp7 = temp5<<3;//queda 38
    unsigned char temp8 = temp5<<4;//queda 70

    unsigned char temp9 = temp>>12;//queda F
    unsigned char temp10 = temp9<<1;//queda 1E 
    unsigned char temp11 = temp9<<3;//queda 78
    unsigned char temp12 = temp9<<4;//queda F0
    
    in = temp ^ temp1 ^ temp2 ^ temp3 ^ temp4 ^ temp5 ^ temp6 ^ temp7 ^ temp8 ^ temp9 ^ temp10 ^ temp11 ^ temp12; 
    return in;
}

__device__ unsigned char multiplicacionENGF2(int caso , unsigned short numero2){
    switch (caso)
    {
    case 1:
        return numero2;
        break;
    case 2:

        numero2 = GF2Redution(numero2 * 2); 

        return numero2 ;
        break;
    case 3:

        numero2 = GF2Redution((numero2*2)^numero2); 

        return numero2;
        break;
    case 9:

        numero2 = GF2Redution((numero2*8)^numero2); 

        return numero2;
        break;

    case 11:


        numero2 = GF2Redution((numero2*8)) ^ GF2Redution((numero2*2)^numero2) ; 

        return numero2;
        break;
    case 13:


        numero2 = GF2Redution((numero2*8)) ^ GF2Redution((numero2*4)) ^ numero2; 

        return numero2;
        break;
    case 14:


        numero2 = GF2Redution((numero2*8)) ^ GF2Redution((numero2*4)) ^ GF2Redution((numero2*2)); 

        return numero2;
        break;

    default:

        break;
    }
    return 0;

}
__device__ void InvAddRoundKey(unsigned int* in, unsigned int * keys, int round){

    unsigned char h[16];
    unsigned char resultado[16];
    int IMC[16]= {
        0x0e, 0x0b, 0x0d, 0x09,
        0x09, 0x0e, 0x0b, 0x0d,
        0x0d, 0x09, 0x0e, 0x0b,
        0x0b, 0x0d, 0x09, 0x0e

    };
    memcpy(h, &keys[round*4], 16);
    for (int i = 0; i < 4; i++) { 
        for (int j = 0; j < 4; j++) { 
            unsigned char temp = 0;
            for(int k = 0; k < 4; k++ ){
                temp= multiplicacionENGF2(IMC[ (j*4)+k ] , h[ (i*4) +3-k ]) ^ temp;
            }
            resultado[ (i*4) +3-j ] = temp;
        }
    }
    unsigned int * matrizXOR;
    matrizXOR = (unsigned int *) resultado;
    for (int i = 0; i < 4; i++) { 
        in[i] = in[i] ^ matrizXOR[i];
    }
}
__device__ void mixColumns(unsigned char* in ){
    for (int i = 0; i < 4; i++){
        unsigned char T1[4]={in[0+i],in[0+i],in[0+i],in[0+i] };
        unsigned char T2[4]={in[4+i],in[4+i],in[4+i],in[4+i] };
        unsigned char T3[4]={in[8+i],in[8+i],in[8+i],in[8+i] };
        unsigned char T4[4]={in[12+i],in[12+i],in[12+i],in[12+i] };
    
        T1[0] =  multiplicacionENGF2(2, T1[0]);
        T1[3] =  multiplicacionENGF2(3, T1[3]);

        T2[0] =  multiplicacionENGF2(3, T2[0]);
        T2[1] =  multiplicacionENGF2(2, T2[1]);

        T3[1] =  multiplicacionENGF2(3, T3[1]);
        T3[2] =  multiplicacionENGF2(2, T3[2]);

        T4[2] =  multiplicacionENGF2(3, T4[2]);
        T4[3] =  multiplicacionENGF2(2, T4[3]);
        
        T1[0] = T1[0] ^ T2[0] ^ T3[0] ^ T4[0];
        T1[1] = T1[1] ^ T2[1] ^ T3[1] ^ T4[1];
        T1[2] = T1[2] ^ T2[2] ^ T3[2] ^ T4[2];
        T1[3] = T1[3] ^ T2[3] ^ T3[3] ^ T4[3];

        in[0+i] = T1[0];  
        in[4+i] = T1[1];
        in[8+i] = T1[2]; 
        in[12+i] = T1[3];
    }
    
}

__device__ void subBytesMixColumns(unsigned int* in, int * T1, int * T2, int * T3, int * T4){
	unsigned char * temp;
    temp = (unsigned char *) in;
    for (int i = 0; i < 16; i=i+4){
        int tempT1 = T1[(int) temp[i+3]];
        int tempT2 = T2[(int) temp[i+2]];
        int tempT3 = T3[(int) temp[i+1]];
        int tempT4 = T4[(int) temp[i] ];

        int tempT5 = tempT1 ^ tempT2 ^ tempT3 ^ tempT4; 
        in[i/4] = tempT5;
    }
}

__device__ void AES_init( unsigned char  *matrizCajaS, int *T1, int *T2, int *T3, int *T4){
    unsigned char matrizCajaSTemp[256]={
        0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,
        0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,
        0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,
        0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,
        0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,
        0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,
        0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,
        0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,
        0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,
        0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,
        0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,
        0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,
        0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
        0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,
        0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,
        0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16
    };
    int T1Temp[256] ={
        0xc66363a5, 0xf87c7c84, 0xee777799, 0xf67b7b8d, 0xfff2f20d, 0xd66b6bbd, 0xde6f6fb1, 0x91c5c554, 0x60303050, 0x2010103, 0xce6767a9, 0x562b2b7d, 0xe7fefe19, 0xb5d7d762, 0x4dababe6, 0xec76769a, 
        0x8fcaca45, 0x1f82829d, 0x89c9c940, 0xfa7d7d87, 0xeffafa15, 0xb25959eb, 0x8e4747c9, 0xfbf0f00b, 0x41adadec, 0xb3d4d467, 0x5fa2a2fd, 0x45afafea, 0x239c9cbf, 0x53a4a4f7, 0xe4727296, 0x9bc0c05b, 
        0x75b7b7c2, 0xe1fdfd1c, 0x3d9393ae, 0x4c26266a, 0x6c36365a, 0x7e3f3f41, 0xf5f7f702, 0x83cccc4f, 0x6834345c, 0x51a5a5f4, 0xd1e5e534, 0xf9f1f108, 0xe2717193, 0xabd8d873, 0x62313153, 0x2a15153f, 
        0x804040c, 0x95c7c752, 0x46232365, 0x9dc3c35e, 0x30181828, 0x379696a1, 0xa05050f, 0x2f9a9ab5, 0xe070709, 0x24121236, 0x1b80809b, 0xdfe2e23d, 0xcdebeb26, 0x4e272769, 0x7fb2b2cd, 0xea75759f, 
        0x1209091b, 0x1d83839e, 0x582c2c74, 0x341a1a2e, 0x361b1b2d, 0xdc6e6eb2, 0xb45a5aee, 0x5ba0a0fb, 0xa45252f6, 0x763b3b4d, 0xb7d6d661, 0x7db3b3ce, 0x5229297b, 0xdde3e33e, 0x5e2f2f71, 0x13848497, 
        0xa65353f5, 0xb9d1d168, 0x0, 0xc1eded2c, 0x40202060, 0xe3fcfc1f, 0x79b1b1c8, 0xb65b5bed, 0xd46a6abe, 0x8dcbcb46, 0x67bebed9, 0x7239394b, 0x944a4ade, 0x984c4cd4, 0xb05858e8, 0x85cfcf4a, 
        0xbbd0d06b, 0xc5efef2a, 0x4faaaae5, 0xedfbfb16, 0x864343c5, 0x9a4d4dd7, 0x66333355, 0x11858594, 0x8a4545cf, 0xe9f9f910, 0x4020206, 0xfe7f7f81, 0xa05050f0, 0x783c3c44, 0x259f9fba, 0x4ba8a8e3, 
        0xa25151f3, 0x5da3a3fe, 0x804040c0, 0x58f8f8a, 0x3f9292ad, 0x219d9dbc, 0x70383848, 0xf1f5f504, 0x63bcbcdf, 0x77b6b6c1, 0xafdada75, 0x42212163, 0x20101030, 0xe5ffff1a, 0xfdf3f30e, 0xbfd2d26d, 
        0x81cdcd4c, 0x180c0c14, 0x26131335, 0xc3ecec2f, 0xbe5f5fe1, 0x359797a2, 0x884444cc, 0x2e171739, 0x93c4c457, 0x55a7a7f2, 0xfc7e7e82, 0x7a3d3d47, 0xc86464ac, 0xba5d5de7, 0x3219192b, 0xe6737395, 
        0xc06060a0, 0x19818198, 0x9e4f4fd1, 0xa3dcdc7f, 0x44222266, 0x542a2a7e, 0x3b9090ab, 0xb888883, 0x8c4646ca, 0xc7eeee29, 0x6bb8b8d3, 0x2814143c, 0xa7dede79, 0xbc5e5ee2, 0x160b0b1d, 0xaddbdb76, 
        0xdbe0e03b, 0x64323256, 0x743a3a4e, 0x140a0a1e, 0x924949db, 0xc06060a, 0x4824246c, 0xb85c5ce4, 0x9fc2c25d, 0xbdd3d36e, 0x43acacef, 0xc46262a6, 0x399191a8, 0x319595a4, 0xd3e4e437, 0xf279798b, 
        0xd5e7e732, 0x8bc8c843, 0x6e373759, 0xda6d6db7, 0x18d8d8c, 0xb1d5d564, 0x9c4e4ed2, 0x49a9a9e0, 0xd86c6cb4, 0xac5656fa, 0xf3f4f407, 0xcfeaea25, 0xca6565af, 0xf47a7a8e, 0x47aeaee9, 0x10080818, 
        0x6fbabad5, 0xf0787888, 0x4a25256f, 0x5c2e2e72, 0x381c1c24, 0x57a6a6f1, 0x73b4b4c7, 0x97c6c651, 0xcbe8e823, 0xa1dddd7c, 0xe874749c, 0x3e1f1f21, 0x964b4bdd, 0x61bdbddc, 0xd8b8b86, 0xf8a8a85, 
        0xe0707090, 0x7c3e3e42, 0x71b5b5c4, 0xcc6666aa, 0x904848d8, 0x6030305, 0xf7f6f601, 0x1c0e0e12, 0xc26161a3, 0x6a35355f, 0xae5757f9, 0x69b9b9d0, 0x17868691, 0x99c1c158, 0x3a1d1d27, 0x279e9eb9, 
        0xd9e1e138, 0xebf8f813, 0x2b9898b3, 0x22111133, 0xd26969bb, 0xa9d9d970, 0x78e8e89, 0x339494a7, 0x2d9b9bb6, 0x3c1e1e22, 0x15878792, 0xc9e9e920, 0x87cece49, 0xaa5555ff, 0x50282878, 0xa5dfdf7a, 
        0x38c8c8f, 0x59a1a1f8, 0x9898980, 0x1a0d0d17, 0x65bfbfda, 0xd7e6e631, 0x844242c6, 0xd06868b8, 0x824141c3, 0x299999b0, 0x5a2d2d77, 0x1e0f0f11, 0x7bb0b0cb, 0xa85454fc, 0x6dbbbbd6, 0x2c16163a
    };
   
    int T2Temp[256] = {
        0xa5c66363, 0x84f87c7c, 0x99ee7777, 0x8df67b7b, 0xdfff2f2, 0xbdd66b6b, 0xb1de6f6f, 0x5491c5c5, 0x50603030, 0x3020101, 0xa9ce6767, 0x7d562b2b, 0x19e7fefe, 0x62b5d7d7, 0xe64dabab, 0x9aec7676, 
        0x458fcaca, 0x9d1f8282, 0x4089c9c9, 0x87fa7d7d, 0x15effafa, 0xebb25959, 0xc98e4747, 0xbfbf0f0, 0xec41adad, 0x67b3d4d4, 0xfd5fa2a2, 0xea45afaf, 0xbf239c9c, 0xf753a4a4, 0x96e47272, 0x5b9bc0c0, 
        0xc275b7b7, 0x1ce1fdfd, 0xae3d9393, 0x6a4c2626, 0x5a6c3636, 0x417e3f3f, 0x2f5f7f7, 0x4f83cccc, 0x5c683434, 0xf451a5a5, 0x34d1e5e5, 0x8f9f1f1, 0x93e27171, 0x73abd8d8, 0x53623131, 0x3f2a1515, 
        0xc080404, 0x5295c7c7, 0x65462323, 0x5e9dc3c3, 0x28301818, 0xa1379696, 0xf0a0505, 0xb52f9a9a, 0x90e0707, 0x36241212, 0x9b1b8080, 0x3ddfe2e2, 0x26cdebeb, 0x694e2727, 0xcd7fb2b2, 0x9fea7575, 
        0x1b120909, 0x9e1d8383, 0x74582c2c, 0x2e341a1a, 0x2d361b1b, 0xb2dc6e6e, 0xeeb45a5a, 0xfb5ba0a0, 0xf6a45252, 0x4d763b3b, 0x61b7d6d6, 0xce7db3b3, 0x7b522929, 0x3edde3e3, 0x715e2f2f, 0x97138484, 
        0xf5a65353, 0x68b9d1d1, 0x0, 0x2cc1eded, 0x60402020, 0x1fe3fcfc, 0xc879b1b1, 0xedb65b5b, 0xbed46a6a, 0x468dcbcb, 0xd967bebe, 0x4b723939, 0xde944a4a, 0xd4984c4c, 0xe8b05858, 0x4a85cfcf, 
        0x6bbbd0d0, 0x2ac5efef, 0xe54faaaa, 0x16edfbfb, 0xc5864343, 0xd79a4d4d, 0x55663333, 0x94118585, 0xcf8a4545, 0x10e9f9f9, 0x6040202, 0x81fe7f7f, 0xf0a05050, 0x44783c3c, 0xba259f9f, 0xe34ba8a8, 
        0xf3a25151, 0xfe5da3a3, 0xc0804040, 0x8a058f8f, 0xad3f9292, 0xbc219d9d, 0x48703838, 0x4f1f5f5, 0xdf63bcbc, 0xc177b6b6, 0x75afdada, 0x63422121, 0x30201010, 0x1ae5ffff, 0xefdf3f3, 0x6dbfd2d2, 
        0x4c81cdcd, 0x14180c0c, 0x35261313, 0x2fc3ecec, 0xe1be5f5f, 0xa2359797, 0xcc884444, 0x392e1717, 0x5793c4c4, 0xf255a7a7, 0x82fc7e7e, 0x477a3d3d, 0xacc86464, 0xe7ba5d5d, 0x2b321919, 0x95e67373, 
        0xa0c06060, 0x98198181, 0xd19e4f4f, 0x7fa3dcdc, 0x66442222, 0x7e542a2a, 0xab3b9090, 0x830b8888, 0xca8c4646, 0x29c7eeee, 0xd36bb8b8, 0x3c281414, 0x79a7dede, 0xe2bc5e5e, 0x1d160b0b, 0x76addbdb, 
        0x3bdbe0e0, 0x56643232, 0x4e743a3a, 0x1e140a0a, 0xdb924949, 0xa0c0606, 0x6c482424, 0xe4b85c5c, 0x5d9fc2c2, 0x6ebdd3d3, 0xef43acac, 0xa6c46262, 0xa8399191, 0xa4319595, 0x37d3e4e4, 0x8bf27979, 
        0x32d5e7e7, 0x438bc8c8, 0x596e3737, 0xb7da6d6d, 0x8c018d8d, 0x64b1d5d5, 0xd29c4e4e, 0xe049a9a9, 0xb4d86c6c, 0xfaac5656, 0x7f3f4f4, 0x25cfeaea, 0xafca6565, 0x8ef47a7a, 0xe947aeae, 0x18100808, 
        0xd56fbaba, 0x88f07878, 0x6f4a2525, 0x725c2e2e, 0x24381c1c, 0xf157a6a6, 0xc773b4b4, 0x5197c6c6, 0x23cbe8e8, 0x7ca1dddd, 0x9ce87474, 0x213e1f1f, 0xdd964b4b, 0xdc61bdbd, 0x860d8b8b, 0x850f8a8a, 
        0x90e07070, 0x427c3e3e, 0xc471b5b5, 0xaacc6666, 0xd8904848, 0x5060303, 0x1f7f6f6, 0x121c0e0e, 0xa3c26161, 0x5f6a3535, 0xf9ae5757, 0xd069b9b9, 0x91178686, 0x5899c1c1, 0x273a1d1d, 0xb9279e9e, 
        0x38d9e1e1, 0x13ebf8f8, 0xb32b9898, 0x33221111, 0xbbd26969, 0x70a9d9d9, 0x89078e8e, 0xa7339494, 0xb62d9b9b, 0x223c1e1e, 0x92158787, 0x20c9e9e9, 0x4987cece, 0xffaa5555, 0x78502828, 0x7aa5dfdf, 
        0x8f038c8c, 0xf859a1a1, 0x80098989, 0x171a0d0d, 0xda65bfbf, 0x31d7e6e6, 0xc6844242, 0xb8d06868, 0xc3824141, 0xb0299999, 0x775a2d2d, 0x111e0f0f, 0xcb7bb0b0, 0xfca85454, 0xd66dbbbb, 0x3a2c1616
    };
    int T3Temp[256] = {
        0x63a5c663, 0x7c84f87c, 0x7799ee77, 0x7b8df67b, 0xf20dfff2, 0x6bbdd66b, 0x6fb1de6f, 0xc55491c5, 0x30506030, 0x1030201, 0x67a9ce67, 0x2b7d562b, 0xfe19e7fe, 0xd762b5d7, 0xabe64dab, 0x769aec76, 
        0xca458fca, 0x829d1f82, 0xc94089c9, 0x7d87fa7d, 0xfa15effa, 0x59ebb259, 0x47c98e47, 0xf00bfbf0, 0xadec41ad, 0xd467b3d4, 0xa2fd5fa2, 0xafea45af, 0x9cbf239c, 0xa4f753a4, 0x7296e472, 0xc05b9bc0, 
        0xb7c275b7, 0xfd1ce1fd, 0x93ae3d93, 0x266a4c26, 0x365a6c36, 0x3f417e3f, 0xf702f5f7, 0xcc4f83cc, 0x345c6834, 0xa5f451a5, 0xe534d1e5, 0xf108f9f1, 0x7193e271, 0xd873abd8, 0x31536231, 0x153f2a15, 
        0x40c0804, 0xc75295c7, 0x23654623, 0xc35e9dc3, 0x18283018, 0x96a13796, 0x50f0a05, 0x9ab52f9a, 0x7090e07, 0x12362412, 0x809b1b80, 0xe23ddfe2, 0xeb26cdeb, 0x27694e27, 0xb2cd7fb2, 0x759fea75, 
        0x91b1209, 0x839e1d83, 0x2c74582c, 0x1a2e341a, 0x1b2d361b, 0x6eb2dc6e, 0x5aeeb45a, 0xa0fb5ba0, 0x52f6a452, 0x3b4d763b, 0xd661b7d6, 0xb3ce7db3, 0x297b5229, 0xe33edde3, 0x2f715e2f, 0x84971384, 
        0x53f5a653, 0xd168b9d1, 0x0, 0xed2cc1ed, 0x20604020, 0xfc1fe3fc, 0xb1c879b1, 0x5bedb65b, 0x6abed46a, 0xcb468dcb, 0xbed967be, 0x394b7239, 0x4ade944a, 0x4cd4984c, 0x58e8b058, 0xcf4a85cf, 
        0xd06bbbd0, 0xef2ac5ef, 0xaae54faa, 0xfb16edfb, 0x43c58643, 0x4dd79a4d, 0x33556633, 0x85941185, 0x45cf8a45, 0xf910e9f9, 0x2060402, 0x7f81fe7f, 0x50f0a050, 0x3c44783c, 0x9fba259f, 0xa8e34ba8, 
        0x51f3a251, 0xa3fe5da3, 0x40c08040, 0x8f8a058f, 0x92ad3f92, 0x9dbc219d, 0x38487038, 0xf504f1f5, 0xbcdf63bc, 0xb6c177b6, 0xda75afda, 0x21634221, 0x10302010, 0xff1ae5ff, 0xf30efdf3, 0xd26dbfd2, 
        0xcd4c81cd, 0xc14180c, 0x13352613, 0xec2fc3ec, 0x5fe1be5f, 0x97a23597, 0x44cc8844, 0x17392e17, 0xc45793c4, 0xa7f255a7, 0x7e82fc7e, 0x3d477a3d, 0x64acc864, 0x5de7ba5d, 0x192b3219, 0x7395e673, 
        0x60a0c060, 0x81981981, 0x4fd19e4f, 0xdc7fa3dc, 0x22664422, 0x2a7e542a, 0x90ab3b90, 0x88830b88, 0x46ca8c46, 0xee29c7ee, 0xb8d36bb8, 0x143c2814, 0xde79a7de, 0x5ee2bc5e, 0xb1d160b, 0xdb76addb, 
        0xe03bdbe0, 0x32566432, 0x3a4e743a, 0xa1e140a, 0x49db9249, 0x60a0c06, 0x246c4824, 0x5ce4b85c, 0xc25d9fc2, 0xd36ebdd3, 0xacef43ac, 0x62a6c462, 0x91a83991, 0x95a43195, 0xe437d3e4, 0x798bf279, 
        0xe732d5e7, 0xc8438bc8, 0x37596e37, 0x6db7da6d, 0x8d8c018d, 0xd564b1d5, 0x4ed29c4e, 0xa9e049a9, 0x6cb4d86c, 0x56faac56, 0xf407f3f4, 0xea25cfea, 0x65afca65, 0x7a8ef47a, 0xaee947ae, 0x8181008, 
        0xbad56fba, 0x7888f078, 0x256f4a25, 0x2e725c2e, 0x1c24381c, 0xa6f157a6, 0xb4c773b4, 0xc65197c6, 0xe823cbe8, 0xdd7ca1dd, 0x749ce874, 0x1f213e1f, 0x4bdd964b, 0xbddc61bd, 0x8b860d8b, 0x8a850f8a, 
        0x7090e070, 0x3e427c3e, 0xb5c471b5, 0x66aacc66, 0x48d89048, 0x3050603, 0xf601f7f6, 0xe121c0e, 0x61a3c261, 0x355f6a35, 0x57f9ae57, 0xb9d069b9, 0x86911786, 0xc15899c1, 0x1d273a1d, 0x9eb9279e, 
        0xe138d9e1, 0xf813ebf8, 0x98b32b98, 0x11332211, 0x69bbd269, 0xd970a9d9, 0x8e89078e, 0x94a73394, 0x9bb62d9b, 0x1e223c1e, 0x87921587, 0xe920c9e9, 0xce4987ce, 0x55ffaa55, 0x28785028, 0xdf7aa5df, 
        0x8c8f038c, 0xa1f859a1, 0x89800989, 0xd171a0d, 0xbfda65bf, 0xe631d7e6, 0x42c68442, 0x68b8d068, 0x41c38241, 0x99b02999, 0x2d775a2d, 0xf111e0f, 0xb0cb7bb0, 0x54fca854, 0xbbd66dbb, 0x163a2c16 
    };
    int T4Temp[256] = {
        0x6363a5c6, 0x7c7c84f8, 0x777799ee, 0x7b7b8df6, 0xf2f20dff, 0x6b6bbdd6, 0x6f6fb1de, 0xc5c55491, 0x30305060, 0x1010302, 0x6767a9ce, 0x2b2b7d56, 0xfefe19e7, 0xd7d762b5, 0xababe64d, 0x76769aec, 
        0xcaca458f, 0x82829d1f, 0xc9c94089, 0x7d7d87fa, 0xfafa15ef, 0x5959ebb2, 0x4747c98e, 0xf0f00bfb, 0xadadec41, 0xd4d467b3, 0xa2a2fd5f, 0xafafea45, 0x9c9cbf23, 0xa4a4f753, 0x727296e4, 0xc0c05b9b, 
        0xb7b7c275, 0xfdfd1ce1, 0x9393ae3d, 0x26266a4c, 0x36365a6c, 0x3f3f417e, 0xf7f702f5, 0xcccc4f83, 0x34345c68, 0xa5a5f451, 0xe5e534d1, 0xf1f108f9, 0x717193e2, 0xd8d873ab, 0x31315362, 0x15153f2a, 
        0x4040c08, 0xc7c75295, 0x23236546, 0xc3c35e9d, 0x18182830, 0x9696a137, 0x5050f0a, 0x9a9ab52f, 0x707090e, 0x12123624, 0x80809b1b, 0xe2e23ddf, 0xebeb26cd, 0x2727694e, 0xb2b2cd7f, 0x75759fea, 
        0x9091b12, 0x83839e1d, 0x2c2c7458, 0x1a1a2e34, 0x1b1b2d36, 0x6e6eb2dc, 0x5a5aeeb4, 0xa0a0fb5b, 0x5252f6a4, 0x3b3b4d76, 0xd6d661b7, 0xb3b3ce7d, 0x29297b52, 0xe3e33edd, 0x2f2f715e, 0x84849713, 
        0x5353f5a6, 0xd1d168b9, 0x0, 0xeded2cc1, 0x20206040, 0xfcfc1fe3, 0xb1b1c879, 0x5b5bedb6, 0x6a6abed4, 0xcbcb468d, 0xbebed967, 0x39394b72, 0x4a4ade94, 0x4c4cd498, 0x5858e8b0, 0xcfcf4a85, 
        0xd0d06bbb, 0xefef2ac5, 0xaaaae54f, 0xfbfb16ed, 0x4343c586, 0x4d4dd79a, 0x33335566, 0x85859411, 0x4545cf8a, 0xf9f910e9, 0x2020604, 0x7f7f81fe, 0x5050f0a0, 0x3c3c4478, 0x9f9fba25, 0xa8a8e34b, 
        0x5151f3a2, 0xa3a3fe5d, 0x4040c080, 0x8f8f8a05, 0x9292ad3f, 0x9d9dbc21, 0x38384870, 0xf5f504f1, 0xbcbcdf63, 0xb6b6c177, 0xdada75af, 0x21216342, 0x10103020, 0xffff1ae5, 0xf3f30efd, 0xd2d26dbf, 
        0xcdcd4c81, 0xc0c1418, 0x13133526, 0xecec2fc3, 0x5f5fe1be, 0x9797a235, 0x4444cc88, 0x1717392e, 0xc4c45793, 0xa7a7f255, 0x7e7e82fc, 0x3d3d477a, 0x6464acc8, 0x5d5de7ba, 0x19192b32, 0x737395e6, 
        0x6060a0c0, 0x81819819, 0x4f4fd19e, 0xdcdc7fa3, 0x22226644, 0x2a2a7e54, 0x9090ab3b, 0x8888830b, 0x4646ca8c, 0xeeee29c7, 0xb8b8d36b, 0x14143c28, 0xdede79a7, 0x5e5ee2bc, 0xb0b1d16, 0xdbdb76ad, 
        0xe0e03bdb, 0x32325664, 0x3a3a4e74, 0xa0a1e14, 0x4949db92, 0x6060a0c, 0x24246c48, 0x5c5ce4b8, 0xc2c25d9f, 0xd3d36ebd, 0xacacef43, 0x6262a6c4, 0x9191a839, 0x9595a431, 0xe4e437d3, 0x79798bf2, 
        0xe7e732d5, 0xc8c8438b, 0x3737596e, 0x6d6db7da, 0x8d8d8c01, 0xd5d564b1, 0x4e4ed29c, 0xa9a9e049, 0x6c6cb4d8, 0x5656faac, 0xf4f407f3, 0xeaea25cf, 0x6565afca, 0x7a7a8ef4, 0xaeaee947, 0x8081810, 
        0xbabad56f, 0x787888f0, 0x25256f4a, 0x2e2e725c, 0x1c1c2438, 0xa6a6f157, 0xb4b4c773, 0xc6c65197, 0xe8e823cb, 0xdddd7ca1, 0x74749ce8, 0x1f1f213e, 0x4b4bdd96, 0xbdbddc61, 0x8b8b860d, 0x8a8a850f, 
        0x707090e0, 0x3e3e427c, 0xb5b5c471, 0x6666aacc, 0x4848d890, 0x3030506, 0xf6f601f7, 0xe0e121c, 0x6161a3c2, 0x35355f6a, 0x5757f9ae, 0xb9b9d069, 0x86869117, 0xc1c15899, 0x1d1d273a, 0x9e9eb927, 
        0xe1e138d9, 0xf8f813eb, 0x9898b32b, 0x11113322, 0x6969bbd2, 0xd9d970a9, 0x8e8e8907, 0x9494a733, 0x9b9bb62d, 0x1e1e223c, 0x87879215, 0xe9e920c9, 0xcece4987, 0x5555ffaa, 0x28287850, 0xdfdf7aa5, 
        0x8c8c8f03, 0xa1a1f859, 0x89898009, 0xd0d171a, 0xbfbfda65, 0xe6e631d7, 0x4242c684, 0x6868b8d0, 0x4141c382, 0x9999b029, 0x2d2d775a, 0xf0f111e, 0xb0b0cb7b, 0x5454fca8, 0xbbbbd66d, 0x16163a2c
    };
    for(int i=0; i<256;i++){
        matrizCajaS[i] = matrizCajaSTemp[i];
        T1[i] = T1Temp[i];
        T2[i] = T2Temp[i];
        T3[i] = T3Temp[i];
        T4[i] = T4Temp[i];
    }
}
__device__ void AES_init_decrypt( unsigned char  *matrizCajaS, int *T1, int *T2, int *T3, int *T4){
    unsigned char matrizCajaSTemp[256]={
        0x52, 0x09, 0x6a, 0xd5, 0x30, 0x36, 0xa5, 0x38, 0xbf, 0x40, 0xa3, 0x9e, 0x81, 0xf3, 0xd7, 0xfb,
        0x7c, 0xe3, 0x39, 0x82, 0x9b, 0x2f, 0xff, 0x87, 0x34, 0x8e, 0x43, 0x44, 0xc4, 0xde, 0xe9, 0xcb,
        0x54, 0x7b, 0x94, 0x32, 0xa6, 0xc2, 0x23, 0x3d, 0xee, 0x4c, 0x95, 0x0b, 0x42, 0xfa, 0xc3, 0x4e,
        0x08, 0x2e, 0xa1, 0x66, 0x28, 0xd9, 0x24, 0xb2, 0x76, 0x5b, 0xa2, 0x49, 0x6d, 0x8b, 0xd1, 0x25,
        0x72, 0xf8, 0xf6, 0x64, 0x86, 0x68, 0x98, 0x16, 0xd4, 0xa4, 0x5c, 0xcc, 0x5d, 0x65, 0xb6, 0x92,
        0x6c, 0x70, 0x48, 0x50, 0xfd, 0xed, 0xb9, 0xda, 0x5e, 0x15, 0x46, 0x57, 0xa7, 0x8d, 0x9d, 0x84,
        0x90, 0xd8, 0xab, 0x00, 0x8c, 0xbc, 0xd3, 0x0a, 0xf7, 0xe4, 0x58, 0x05, 0xb8, 0xb3, 0x45, 0x06,
        0xd0, 0x2c, 0x1e, 0x8f, 0xca, 0x3f, 0x0f, 0x02, 0xc1, 0xaf, 0xbd, 0x03, 0x01, 0x13, 0x8a, 0x6b,
        0x3a, 0x91, 0x11, 0x41, 0x4f, 0x67, 0xdc, 0xea, 0x97, 0xf2, 0xcf, 0xce, 0xf0, 0xb4, 0xe6, 0x73,
        0x96, 0xac, 0x74, 0x22, 0xe7, 0xad, 0x35, 0x85, 0xe2, 0xf9, 0x37, 0xe8, 0x1c, 0x75, 0xdf, 0x6e,
        0x47, 0xf1, 0x1a, 0x71, 0x1d, 0x29, 0xc5, 0x89, 0x6f, 0xb7, 0x62, 0x0e, 0xaa, 0x18, 0xbe, 0x1b,
        0xfc, 0x56, 0x3e, 0x4b, 0xc6, 0xd2, 0x79, 0x20, 0x9a, 0xdb, 0xc0, 0xfe, 0x78, 0xcd, 0x5a, 0xf4,
        0x1f, 0xdd, 0xa8, 0x33, 0x88, 0x07, 0xc7, 0x31, 0xb1, 0x12, 0x10, 0x59, 0x27, 0x80, 0xec, 0x5f,
        0x60, 0x51, 0x7f, 0xa9, 0x19, 0xb5, 0x4a, 0x0d, 0x2d, 0xe5, 0x7a, 0x9f, 0x93, 0xc9, 0x9c, 0xef,
        0xa0, 0xe0, 0x3b, 0x4d, 0xae, 0x2a, 0xf5, 0xb0, 0xc8, 0xeb, 0xbb, 0x3c, 0x83, 0x53, 0x99, 0x61,
        0x17, 0x2b, 0x04, 0x7e, 0xba, 0x77, 0xd6, 0x26, 0xe1, 0x69, 0x14, 0x63, 0x55, 0x21, 0x0c, 0x7d
    };
    int T1Temp[256] ={
        0x51f4a750, 0x7e416553, 0x1a17a4c3, 0x3a275e96, 0x3bab6bcb, 0x1f9d45f1, 0xacfa58ab, 0x4be30393, 0x2030fa55, 0xad766df6, 0x88cc7691, 0xf5024c25, 0x4fe5d7fc, 0xc52acbd7, 0x26354480, 0xb562a38f, 
        0xdeb15a49, 0x25ba1b67, 0x45ea0e98, 0x5dfec0e1, 0xc32f7502, 0x814cf012, 0x8d4697a3, 0x6bd3f9c6, 0x38f5fe7, 0x15929c95, 0xbf6d7aeb, 0x955259da, 0xd4be832d, 0x587421d3, 0x49e06929, 0x8ec9c844, 
        0x75c2896a, 0xf48e7978, 0x99583e6b, 0x27b971dd, 0xbee14fb6, 0xf088ad17, 0xc920ac66, 0x7dce3ab4, 0x63df4a18, 0xe51a3182, 0x97513360, 0x62537f45, 0xb16477e0, 0xbb6bae84, 0xfe81a01c, 0xf9082b94, 
        0x70486858, 0x8f45fd19, 0x94de6c87, 0x527bf8b7, 0xab73d323, 0x724b02e2, 0xe31f8f57, 0x6655ab2a, 0xb2eb2807, 0x2fb5c203, 0x86c57b9a, 0xd33708a5, 0x302887f2, 0x23bfa5b2, 0x2036aba, 0xed16825c, 
        0x8acf1c2b, 0xa779b492, 0xf307f2f0, 0x4e69e2a1, 0x65daf4cd, 0x605bed5, 0xd134621f, 0xc4a6fe8a, 0x342e539d, 0xa2f355a0, 0x58ae132, 0xa4f6eb75, 0xb83ec39, 0x4060efaa, 0x5e719f06, 0xbd6e1051, 
        0x3e218af9, 0x96dd063d, 0xdd3e05ae, 0x4de6bd46, 0x91548db5, 0x71c45d05, 0x406d46f, 0x605015ff, 0x1998fb24, 0xd6bde997, 0x894043cc, 0x67d99e77, 0xb0e842bd, 0x7898b88, 0xe7195b38, 0x79c8eedb, 
        0xa17c0a47, 0x7c420fe9, 0xf8841ec9, 0x0, 0x9808683, 0x322bed48, 0x1e1170ac, 0x6c5a724e, 0xfd0efffb, 0xf853856, 0x3daed51e, 0x362d3927, 0xa0fd964, 0x685ca621, 0x9b5b54d1, 0x24362e3a, 
        0xc0a67b1, 0x9357e70f, 0xb4ee96d2, 0x1b9b919e, 0x80c0c54f, 0x61dc20a2, 0x5a774b69, 0x1c121a16, 0xe293ba0a, 0xc0a02ae5, 0x3c22e043, 0x121b171d, 0xe090d0b, 0xf28bc7ad, 0x2db6a8b9, 0x141ea9c8, 
        0x57f11985, 0xaf75074c, 0xee99ddbb, 0xa37f60fd, 0xf701269f, 0x5c72f5bc, 0x44663bc5, 0x5bfb7e34, 0x8b432976, 0xcb23c6dc, 0xb6edfc68, 0xb8e4f163, 0xd731dcca, 0x42638510, 0x13972240, 0x84c61120, 
        0x854a247d, 0xd2bb3df8, 0xaef93211, 0xc729a16d, 0x1d9e2f4b, 0xdcb230f3, 0xd8652ec, 0x77c1e3d0, 0x2bb3166c, 0xa970b999, 0x119448fa, 0x47e96422, 0xa8fc8cc4, 0xa0f03f1a, 0x567d2cd8, 0x223390ef, 
        0x87494ec7, 0xd938d1c1, 0x8ccaa2fe, 0x98d40b36, 0xa6f581cf, 0xa57ade28, 0xdab78e26, 0x3fadbfa4, 0x2c3a9de4, 0x5078920d, 0x6a5fcc9b, 0x547e4662, 0xf68d13c2, 0x90d8b8e8, 0x2e39f75e, 0x82c3aff5, 
        0x9f5d80be, 0x69d0937c, 0x6fd52da9, 0xcf2512b3, 0xc8ac993b, 0x10187da7, 0xe89c636e, 0xdb3bbb7b, 0xcd267809, 0x6e5918f4, 0xec9ab701, 0x834f9aa8, 0xe6956e65, 0xaaffe67e, 0x21bccf08, 0xef15e8e6, 
        0xbae79bd9, 0x4a6f36ce, 0xea9f09d4, 0x29b07cd6, 0x31a4b2af, 0x2a3f2331, 0xc6a59430, 0x35a266c0, 0x744ebc37, 0xfc82caa6, 0xe090d0b0, 0x33a7d815, 0xf104984a, 0x41ecdaf7, 0x7fcd500e, 0x1791f62f, 
        0x764dd68d, 0x43efb04d, 0xccaa4d54, 0xe49604df, 0x9ed1b5e3, 0x4c6a881b, 0xc12c1fb8, 0x4665517f, 0x9d5eea04, 0x18c355d, 0xfa877473, 0xfb0b412e, 0xb3671d5a, 0x92dbd252, 0xe9105633, 0x6dd64713, 
        0x9ad7618c, 0x37a10c7a, 0x59f8148e, 0xeb133c89, 0xcea927ee, 0xb761c935, 0xe11ce5ed, 0x7a47b13c, 0x9cd2df59, 0x55f2733f, 0x1814ce79, 0x73c737bf, 0x53f7cdea, 0x5ffdaa5b, 0xdf3d6f14, 0x7844db86, 
        0xcaaff381, 0xb968c43e, 0x3824342c, 0xc2a3405f, 0x161dc372, 0xbce2250c, 0x283c498b, 0xff0d9541, 0x39a80171, 0x80cb3de, 0xd8b4e49c, 0x6456c190, 0x7bcb8461, 0xd532b670, 0x486c5c74, 0xd0b85742
    };
   
    int T2Temp[256] = {
        0x5051f4a7, 0x537e4165, 0xc31a17a4, 0x963a275e, 0xcb3bab6b, 0xf11f9d45, 0xabacfa58, 0x934be303, 0x552030fa, 0xf6ad766d, 0x9188cc76, 0x25f5024c, 0xfc4fe5d7, 0xd7c52acb, 0x80263544, 0x8fb562a3, 
        0x49deb15a, 0x6725ba1b, 0x9845ea0e, 0xe15dfec0, 0x2c32f75, 0x12814cf0, 0xa38d4697, 0xc66bd3f9, 0xe7038f5f, 0x9515929c, 0xebbf6d7a, 0xda955259, 0x2dd4be83, 0xd3587421, 0x2949e069, 0x448ec9c8, 
        0x6a75c289, 0x78f48e79, 0x6b99583e, 0xdd27b971, 0xb6bee14f, 0x17f088ad, 0x66c920ac, 0xb47dce3a, 0x1863df4a, 0x82e51a31, 0x60975133, 0x4562537f, 0xe0b16477, 0x84bb6bae, 0x1cfe81a0, 0x94f9082b, 
        0x58704868, 0x198f45fd, 0x8794de6c, 0xb7527bf8, 0x23ab73d3, 0xe2724b02, 0x57e31f8f, 0x2a6655ab, 0x7b2eb28, 0x32fb5c2, 0x9a86c57b, 0xa5d33708, 0xf2302887, 0xb223bfa5, 0xba02036a, 0x5ced1682, 
        0x2b8acf1c, 0x92a779b4, 0xf0f307f2, 0xa14e69e2, 0xcd65daf4, 0xd50605be, 0x1fd13462, 0x8ac4a6fe, 0x9d342e53, 0xa0a2f355, 0x32058ae1, 0x75a4f6eb, 0x390b83ec, 0xaa4060ef, 0x65e719f, 0x51bd6e10, 
        0xf93e218a, 0x3d96dd06, 0xaedd3e05, 0x464de6bd, 0xb591548d, 0x571c45d, 0x6f0406d4, 0xff605015, 0x241998fb, 0x97d6bde9, 0xcc894043, 0x7767d99e, 0xbdb0e842, 0x8807898b, 0x38e7195b, 0xdb79c8ee, 
        0x47a17c0a, 0xe97c420f, 0xc9f8841e, 0x0, 0x83098086, 0x48322bed, 0xac1e1170, 0x4e6c5a72, 0xfbfd0eff, 0x560f8538, 0x1e3daed5, 0x27362d39, 0x640a0fd9, 0x21685ca6, 0xd19b5b54, 0x3a24362e, 
        0xb10c0a67, 0xf9357e7, 0xd2b4ee96, 0x9e1b9b91, 0x4f80c0c5, 0xa261dc20, 0x695a774b, 0x161c121a, 0xae293ba, 0xe5c0a02a, 0x433c22e0, 0x1d121b17, 0xb0e090d, 0xadf28bc7, 0xb92db6a8, 0xc8141ea9, 
        0x8557f119, 0x4caf7507, 0xbbee99dd, 0xfda37f60, 0x9ff70126, 0xbc5c72f5, 0xc544663b, 0x345bfb7e, 0x768b4329, 0xdccb23c6, 0x68b6edfc, 0x63b8e4f1, 0xcad731dc, 0x10426385, 0x40139722, 0x2084c611, 
        0x7d854a24, 0xf8d2bb3d, 0x11aef932, 0x6dc729a1, 0x4b1d9e2f, 0xf3dcb230, 0xec0d8652, 0xd077c1e3, 0x6c2bb316, 0x99a970b9, 0xfa119448, 0x2247e964, 0xc4a8fc8c, 0x1aa0f03f, 0xd8567d2c, 0xef223390, 
        0xc787494e, 0xc1d938d1, 0xfe8ccaa2, 0x3698d40b, 0xcfa6f581, 0x28a57ade, 0x26dab78e, 0xa43fadbf, 0xe42c3a9d, 0xd507892, 0x9b6a5fcc, 0x62547e46, 0xc2f68d13, 0xe890d8b8, 0x5e2e39f7, 0xf582c3af, 
        0xbe9f5d80, 0x7c69d093, 0xa96fd52d, 0xb3cf2512, 0x3bc8ac99, 0xa710187d, 0x6ee89c63, 0x7bdb3bbb, 0x9cd2678, 0xf46e5918, 0x1ec9ab7, 0xa8834f9a, 0x65e6956e, 0x7eaaffe6, 0x821bccf, 0xe6ef15e8, 
        0xd9bae79b, 0xce4a6f36, 0xd4ea9f09, 0xd629b07c, 0xaf31a4b2, 0x312a3f23, 0x30c6a594, 0xc035a266, 0x37744ebc, 0xa6fc82ca, 0xb0e090d0, 0x1533a7d8, 0x4af10498, 0xf741ecda, 0xe7fcd50, 0x2f1791f6, 
        0x8d764dd6, 0x4d43efb0, 0x54ccaa4d, 0xdfe49604, 0xe39ed1b5, 0x1b4c6a88, 0xb8c12c1f, 0x7f466551, 0x49d5eea, 0x5d018c35, 0x73fa8774, 0x2efb0b41, 0x5ab3671d, 0x5292dbd2, 0x33e91056, 0x136dd647, 
        0x8c9ad761, 0x7a37a10c, 0x8e59f814, 0x89eb133c, 0xeecea927, 0x35b761c9, 0xede11ce5, 0x3c7a47b1, 0x599cd2df, 0x3f55f273, 0x791814ce, 0xbf73c737, 0xea53f7cd, 0x5b5ffdaa, 0x14df3d6f, 0x867844db, 
        0x81caaff3, 0x3eb968c4, 0x2c382434, 0x5fc2a340, 0x72161dc3, 0xcbce225, 0x8b283c49, 0x41ff0d95, 0x7139a801, 0xde080cb3, 0x9cd8b4e4, 0x906456c1, 0x617bcb84, 0x70d532b6, 0x74486c5c, 0x42d0b857
    };
    int T3Temp[256] = {
        0xa75051f4, 0x65537e41, 0xa4c31a17, 0x5e963a27, 0x6bcb3bab, 0x45f11f9d, 0x58abacfa, 0x3934be3, 0xfa552030, 0x6df6ad76, 0x769188cc, 0x4c25f502, 0xd7fc4fe5, 0xcbd7c52a, 0x44802635, 0xa38fb562, 
        0x5a49deb1, 0x1b6725ba, 0xe9845ea, 0xc0e15dfe, 0x7502c32f, 0xf012814c, 0x97a38d46, 0xf9c66bd3, 0x5fe7038f, 0x9c951592, 0x7aebbf6d, 0x59da9552, 0x832dd4be, 0x21d35874, 0x692949e0, 0xc8448ec9, 
        0x896a75c2, 0x7978f48e, 0x3e6b9958, 0x71dd27b9, 0x4fb6bee1, 0xad17f088, 0xac66c920, 0x3ab47dce, 0x4a1863df, 0x3182e51a, 0x33609751, 0x7f456253, 0x77e0b164, 0xae84bb6b, 0xa01cfe81, 0x2b94f908, 
        0x68587048, 0xfd198f45, 0x6c8794de, 0xf8b7527b, 0xd323ab73, 0x2e2724b, 0x8f57e31f, 0xab2a6655, 0x2807b2eb, 0xc2032fb5, 0x7b9a86c5, 0x8a5d337, 0x87f23028, 0xa5b223bf, 0x6aba0203, 0x825ced16, 
        0x1c2b8acf, 0xb492a779, 0xf2f0f307, 0xe2a14e69, 0xf4cd65da, 0xbed50605, 0x621fd134, 0xfe8ac4a6, 0x539d342e, 0x55a0a2f3, 0xe132058a, 0xeb75a4f6, 0xec390b83, 0xefaa4060, 0x9f065e71, 0x1051bd6e, 
        0x8af93e21, 0x63d96dd, 0x5aedd3e, 0xbd464de6, 0x8db59154, 0x5d0571c4, 0xd46f0406, 0x15ff6050, 0xfb241998, 0xe997d6bd, 0x43cc8940, 0x9e7767d9, 0x42bdb0e8, 0x8b880789, 0x5b38e719, 0xeedb79c8, 
        0xa47a17c, 0xfe97c42, 0x1ec9f884, 0x0, 0x86830980, 0xed48322b, 0x70ac1e11, 0x724e6c5a, 0xfffbfd0e, 0x38560f85, 0xd51e3dae, 0x3927362d, 0xd9640a0f, 0xa621685c, 0x54d19b5b, 0x2e3a2436, 
        0x67b10c0a, 0xe70f9357, 0x96d2b4ee, 0x919e1b9b, 0xc54f80c0, 0x20a261dc, 0x4b695a77, 0x1a161c12, 0xba0ae293, 0x2ae5c0a0, 0xe0433c22, 0x171d121b, 0xd0b0e09, 0xc7adf28b, 0xa8b92db6, 0xa9c8141e, 
        0x198557f1, 0x74caf75, 0xddbbee99, 0x60fda37f, 0x269ff701, 0xf5bc5c72, 0x3bc54466, 0x7e345bfb, 0x29768b43, 0xc6dccb23, 0xfc68b6ed, 0xf163b8e4, 0xdccad731, 0x85104263, 0x22401397, 0x112084c6, 
        0x247d854a, 0x3df8d2bb, 0x3211aef9, 0xa16dc729, 0x2f4b1d9e, 0x30f3dcb2, 0x52ec0d86, 0xe3d077c1, 0x166c2bb3, 0xb999a970, 0x48fa1194, 0x642247e9, 0x8cc4a8fc, 0x3f1aa0f0, 0x2cd8567d, 0x90ef2233, 
        0x4ec78749, 0xd1c1d938, 0xa2fe8cca, 0xb3698d4, 0x81cfa6f5, 0xde28a57a, 0x8e26dab7, 0xbfa43fad, 0x9de42c3a, 0x920d5078, 0xcc9b6a5f, 0x4662547e, 0x13c2f68d, 0xb8e890d8, 0xf75e2e39, 0xaff582c3, 
        0x80be9f5d, 0x937c69d0, 0x2da96fd5, 0x12b3cf25, 0x993bc8ac, 0x7da71018, 0x636ee89c, 0xbb7bdb3b, 0x7809cd26, 0x18f46e59, 0xb701ec9a, 0x9aa8834f, 0x6e65e695, 0xe67eaaff, 0xcf0821bc, 0xe8e6ef15, 
        0x9bd9bae7, 0x36ce4a6f, 0x9d4ea9f, 0x7cd629b0, 0xb2af31a4, 0x23312a3f, 0x9430c6a5, 0x66c035a2, 0xbc37744e, 0xcaa6fc82, 0xd0b0e090, 0xd81533a7, 0x984af104, 0xdaf741ec, 0x500e7fcd, 0xf62f1791, 
        0xd68d764d, 0xb04d43ef, 0x4d54ccaa, 0x4dfe496, 0xb5e39ed1, 0x881b4c6a, 0x1fb8c12c, 0x517f4665, 0xea049d5e, 0x355d018c, 0x7473fa87, 0x412efb0b, 0x1d5ab367, 0xd25292db, 0x5633e910, 0x47136dd6, 
        0x618c9ad7, 0xc7a37a1, 0x148e59f8, 0x3c89eb13, 0x27eecea9, 0xc935b761, 0xe5ede11c, 0xb13c7a47, 0xdf599cd2, 0x733f55f2, 0xce791814, 0x37bf73c7, 0xcdea53f7, 0xaa5b5ffd, 0x6f14df3d, 0xdb867844, 
        0xf381caaf, 0xc43eb968, 0x342c3824, 0x405fc2a3, 0xc372161d, 0x250cbce2, 0x498b283c, 0x9541ff0d, 0x17139a8, 0xb3de080c, 0xe49cd8b4, 0xc1906456, 0x84617bcb, 0xb670d532, 0x5c74486c, 0x5742d0b8
    };
    int T4Temp[256] = {
        0xf4a75051, 0x4165537e, 0x17a4c31a, 0x275e963a, 0xab6bcb3b, 0x9d45f11f, 0xfa58abac, 0xe303934b, 0x30fa5520, 0x766df6ad, 0xcc769188, 0x24c25f5, 0xe5d7fc4f, 0x2acbd7c5, 0x35448026, 0x62a38fb5, 
        0xb15a49de, 0xba1b6725, 0xea0e9845, 0xfec0e15d, 0x2f7502c3, 0x4cf01281, 0x4697a38d, 0xd3f9c66b, 0x8f5fe703, 0x929c9515, 0x6d7aebbf, 0x5259da95, 0xbe832dd4, 0x7421d358, 0xe0692949, 0xc9c8448e, 
        0xc2896a75, 0x8e7978f4, 0x583e6b99, 0xb971dd27, 0xe14fb6be, 0x88ad17f0, 0x20ac66c9, 0xce3ab47d, 0xdf4a1863, 0x1a3182e5, 0x51336097, 0x537f4562, 0x6477e0b1, 0x6bae84bb, 0x81a01cfe, 0x82b94f9, 
        0x48685870, 0x45fd198f, 0xde6c8794, 0x7bf8b752, 0x73d323ab, 0x4b02e272, 0x1f8f57e3, 0x55ab2a66, 0xeb2807b2, 0xb5c2032f, 0xc57b9a86, 0x3708a5d3, 0x2887f230, 0xbfa5b223, 0x36aba02, 0x16825ced, 
        0xcf1c2b8a, 0x79b492a7, 0x7f2f0f3, 0x69e2a14e, 0xdaf4cd65, 0x5bed506, 0x34621fd1, 0xa6fe8ac4, 0x2e539d34, 0xf355a0a2, 0x8ae13205, 0xf6eb75a4, 0x83ec390b, 0x60efaa40, 0x719f065e, 0x6e1051bd, 
        0x218af93e, 0xdd063d96, 0x3e05aedd, 0xe6bd464d, 0x548db591, 0xc45d0571, 0x6d46f04, 0x5015ff60, 0x98fb2419, 0xbde997d6, 0x4043cc89, 0xd99e7767, 0xe842bdb0, 0x898b8807, 0x195b38e7, 0xc8eedb79, 
        0x7c0a47a1, 0x420fe97c, 0x841ec9f8, 0x0, 0x80868309, 0x2bed4832, 0x1170ac1e, 0x5a724e6c, 0xefffbfd, 0x8538560f, 0xaed51e3d, 0x2d392736, 0xfd9640a, 0x5ca62168, 0x5b54d19b, 0x362e3a24, 
        0xa67b10c, 0x57e70f93, 0xee96d2b4, 0x9b919e1b, 0xc0c54f80, 0xdc20a261, 0x774b695a, 0x121a161c, 0x93ba0ae2, 0xa02ae5c0, 0x22e0433c, 0x1b171d12, 0x90d0b0e, 0x8bc7adf2, 0xb6a8b92d, 0x1ea9c814, 
        0xf1198557, 0x75074caf, 0x99ddbbee, 0x7f60fda3, 0x1269ff7, 0x72f5bc5c, 0x663bc544, 0xfb7e345b, 0x4329768b, 0x23c6dccb, 0xedfc68b6, 0xe4f163b8, 0x31dccad7, 0x63851042, 0x97224013, 0xc6112084, 
        0x4a247d85, 0xbb3df8d2, 0xf93211ae, 0x29a16dc7, 0x9e2f4b1d, 0xb230f3dc, 0x8652ec0d, 0xc1e3d077, 0xb3166c2b, 0x70b999a9, 0x9448fa11, 0xe9642247, 0xfc8cc4a8, 0xf03f1aa0, 0x7d2cd856, 0x3390ef22, 
        0x494ec787, 0x38d1c1d9, 0xcaa2fe8c, 0xd40b3698, 0xf581cfa6, 0x7ade28a5, 0xb78e26da, 0xadbfa43f, 0x3a9de42c, 0x78920d50, 0x5fcc9b6a, 0x7e466254, 0x8d13c2f6, 0xd8b8e890, 0x39f75e2e, 0xc3aff582, 
        0x5d80be9f, 0xd0937c69, 0xd52da96f, 0x2512b3cf, 0xac993bc8, 0x187da710, 0x9c636ee8, 0x3bbb7bdb, 0x267809cd, 0x5918f46e, 0x9ab701ec, 0x4f9aa883, 0x956e65e6, 0xffe67eaa, 0xbccf0821, 0x15e8e6ef, 
        0xe79bd9ba, 0x6f36ce4a, 0x9f09d4ea, 0xb07cd629, 0xa4b2af31, 0x3f23312a, 0xa59430c6, 0xa266c035, 0x4ebc3774, 0x82caa6fc, 0x90d0b0e0, 0xa7d81533, 0x4984af1, 0xecdaf741, 0xcd500e7f, 0x91f62f17, 
        0x4dd68d76, 0xefb04d43, 0xaa4d54cc, 0x9604dfe4, 0xd1b5e39e, 0x6a881b4c, 0x2c1fb8c1, 0x65517f46, 0x5eea049d, 0x8c355d01, 0x877473fa, 0xb412efb, 0x671d5ab3, 0xdbd25292, 0x105633e9, 0xd647136d, 
        0xd7618c9a, 0xa10c7a37, 0xf8148e59, 0x133c89eb, 0xa927eece, 0x61c935b7, 0x1ce5ede1, 0x47b13c7a, 0xd2df599c, 0xf2733f55, 0x14ce7918, 0xc737bf73, 0xf7cdea53, 0xfdaa5b5f, 0x3d6f14df, 0x44db8678, 
        0xaff381ca, 0x68c43eb9, 0x24342c38, 0xa3405fc2, 0x1dc37216, 0xe2250cbc, 0x3c498b28, 0xd9541ff, 0xa8017139, 0xcb3de08, 0xb4e49cd8, 0x56c19064, 0xcb84617b, 0x32b670d5, 0x6c5c7448, 0xb85742d0
    };
    for(int i=0; i<256;i++){
        matrizCajaS[i] = matrizCajaSTemp[i];
        T1[i] = T1Temp[i];
        T2[i] = T2Temp[i];
        T3[i] = T3Temp[i];
        T4[i] = T4Temp[i];
    }
}
__device__ void AES_128(aesBlock *m, unsigned int *keys, int index){
    __shared__ unsigned char matrizCajaS[256];
    __shared__ int T1[256]; 
    __shared__ int T2[256]; 
    __shared__ int T3[256]; 
    __shared__ int T4[256];
        if(threadIdx.x == 0 || index == 0){
            AES_init(matrizCajaS, T1, T2, T3, T4);
        }
        
        int shifttab[16]= {
            12, 9, 6, 3,   
            0, 13, 10, 7,  
            4, 1, 14, 11,
            8, 5, 2, 15 
            };
        
        // __syncthreads(); preguntar porque esto hace que nada salga
        unsigned int block[4];

        for (int i = 0 ; i< 4 ; i++){
            block[i]= m[index].block[i];
        }
        
        addRoundKey( block, keys,0);
        for (int j = 1; j < 10; j++){
            // subBytes(block, matrizCajaS);
            shiftRows(block, shifttab);
            //mixColumns(block);
            subBytesMixColumns(block,  T1,  T2,  T3,  T4);

            addRoundKey( block, keys,j); // 
        }
        subBytes(block, matrizCajaS);
        shiftRows(block, shifttab);
        addRoundKey( block, keys,10);
        for (int i = 0 ; i< 4 ; i++){
            m[index].block[i]= block[i];
        }
}

__device__ void OCBAESDelta1Rounds(unsigned int block[4],  unsigned int *keys){
        unsigned char matrizCajaS[256];
        int T1[256]; 
        int T2[256]; 
        int T3[256]; 
        int T4[256];
            AES_init(matrizCajaS, T1, T2, T3, T4);
        int shifttab[16]= {
            12, 9, 6, 3,   
            0, 13, 10, 7,  
            4, 1, 14, 11,
            8, 5, 2, 15 
            };
        addRoundKey( block, keys,0);
        for (int j = 1; j < 2; j++){
            shiftRows(block, shifttab);
            subBytesMixColumns(block,  T1,  T2,  T3,  T4);
            addRoundKey( block, keys,j); // 
        }
        subBytes(block, matrizCajaS);
        shiftRows(block, shifttab);
        addRoundKey( block, keys,10);
}

__device__ void AES_128Decrypt(aesBlock *m, unsigned int *keys, int index ){
    __shared__ unsigned char matrizCajaS[256];
    __shared__ int T1[256]; 
    __shared__ int T2[256]; 
    __shared__ int T3[256]; 
    __shared__ int T4[256];
        if(threadIdx.x == 0 || index == 0){
            AES_init_decrypt(matrizCajaS, T1, T2, T3, T4);
        }
        
        int shifttab[16]= {
            4, 9, 14, 3,   
            8, 13, 2, 7,  
            12, 1, 6, 11,
            0, 5, 10, 15 
            };
        
        __syncthreads();
        unsigned int block[4];

        for (int i = 0 ; i< 4 ; i++){
            block[i]= m[index].block[i];
        }
        
        addRoundKey( block, keys,10);
        for (int j = 9; j > 0; j--){
            // subBytes(block, matrizCajaS);
            shiftRows(block, shifttab);
            // mixColumns(block);
            subBytesMixColumns(block,  T1,  T2,  T3,  T4);

            InvAddRoundKey(block, keys,j);

        }
        subBytes(block, matrizCajaS);
        shiftRows(block, shifttab);
        addRoundKey( block, keys,0);
        for (int i = 0 ; i< 4 ; i++){
            m[index].block[i]= block[i];
        }

}
__global__ void Encrypt(aesBlock *m, unsigned long long mlen, unsigned int *keys){
    int index = blockDim.x*blockIdx.x + threadIdx.x;
    if( index<mlen/16){
        AES_128(m, keys,index);
    }
}

void AES128Encrypt(aesBlock *m, unsigned long long mlen, unsigned int *keys){
    aesBlock *mCuda;
    unsigned int *keysCuda;

    int sizeMessage = (mlen/16)*sizeof(class aesBlock);
    int sizeKeys = 11*4*sizeof(unsigned int);

    hipMalloc(&mCuda, sizeMessage);
    hipMalloc(&keysCuda, sizeKeys);

    hipMemcpy(mCuda,m,sizeMessage,hipMemcpyDefault);
    hipMemcpy(keysCuda,keys,sizeKeys,hipMemcpyDefault);

    dim3 nb( (unsigned int) ceil((double) mlen/16.0) ) ; 
    dim3 nt(2); 

    Encrypt<<<nb, nt>>>(mCuda, mlen,keysCuda);
    hipMemcpy(m, mCuda, sizeMessage, hipMemcpyDefault);

    hipFree(mCuda); hipFree(keysCuda);  
}


__global__ void OCB128EncryptRandomAcces(aesBlock *m,aesBlock *delta, aesBlock *S,unsigned long long mlen,unsigned long long mlen2, unsigned long long deltalen, unsigned int *keys){
    int index = blockDim.x*blockIdx.x + threadIdx.x;
    unsigned long long deltaIndex = floor( (double) index/64);//esto se cambia para aes 1 ronda

    if( index<mlen2/16){

        __syncthreads();
        unsigned int deltaBlock[4];
        //ultimo bloque
        if(index == (mlen2-16)/16){
            for (int i = 0 ; i< 4 ; i++){
                deltaBlock[i]= delta[deltaIndex].block[i]+index+3;
            }
            OCBAESDelta1Rounds(deltaBlock, keys);
    
            XOR_128(m[index].block,deltaBlock);

            AES_128(m, keys,index);
    
            XOR_128(m[index].block,S[0].block);
        }
        //penultimo bloque
        else if(index == ((mlen2-32)/16) ){
            if(mlen%16==0){
                for (int i = 0 ; i< 4 ; i++){
                    deltaBlock[i]= delta[deltaIndex].block[i]+index;
                }
        
                OCBAESDelta1Rounds(deltaBlock, keys);
        
                XOR_128(m[index].block,deltaBlock);
        
                AES_128(m, keys,index);
        
                XOR_128(m[index].block,deltaBlock);
            }else{
                aesBlock *aestemp;
                aestemp = new aesBlock [1];
                for (int i = 0 ; i< 4 ; i++){
                    aestemp[0].block[i] = delta[deltaIndex].block[i]+index+1;
                }
                OCBAESDelta1Rounds(aestemp[0].block,keys);

                AES_128(aestemp, keys,0);

                XOR_128(m[index].block,aestemp[0].block);

                for (int i = 0 ; i< 4 ; i++){
                    aestemp[0].block[i] = 0;
                }
                memcpy(aestemp[0].block, m[index].block, mlen);
                memcpy(m[index].block, aestemp[0].block, 16);
            }
        }
        else{
            for (int i = 0 ; i< 4 ; i++){
                deltaBlock[i]= delta[deltaIndex].block[i]+index;
            }
    
            OCBAESDelta1Rounds(deltaBlock, keys);
    
            XOR_128(m[index].block,deltaBlock);
    
            AES_128(m, keys,index);
    
            XOR_128(m[index].block,deltaBlock);
        }

    }
}


__global__ void OCB128EncryptRandomAccesAsociatedData(aesBlock *ad,aesBlock *delta, aesBlock *Ek1, unsigned long long adlen, unsigned long long adlenMultiplo16, unsigned long long deltalen, unsigned int *keys){
    int index = blockDim.x*blockIdx.x + threadIdx.x;
    unsigned long long deltaIndex = floor( (double) index/64);//esto se cambia para aes 1 ronda

    if( index< (adlenMultiplo16/16)){
        // if( index==0){

        
        __syncthreads();
        unsigned int deltaBlock[4];
        
        for (int i = 0 ; i< 4 ; i++){
            deltaBlock[i]= delta[deltaIndex].block[i]+index;
        }

        if(index == ((adlenMultiplo16-16)/16) && adlen%16!=0 ){
            for (int i = 0 ; i< 4 ; i++){
                deltaBlock[i]= delta[deltaIndex].block[i]+index+1;
            }
            //calculo lamda 5
            OCBAESDelta1Rounds(deltaBlock, keys);
            XOR_128(deltaBlock,Ek1[0].block);
            //fin calculo lamda 5

            //Al XOR lamba5
            XOR_128(ad[index].block,deltaBlock);

            AES_128(ad, keys,index);
            // imprimiArregloCudaInt(16,ad[index].block);

        }else{
            //calculo lamda 5
            OCBAESDelta1Rounds(deltaBlock, keys);
            XOR_128(deltaBlock,Ek1[0].block);
            //fin calculo lamda 5

            //Al XOR lamba5
            XOR_128(ad[index].block,deltaBlock);

            AES_128(ad, keys,index);
            // imprimiArregloCudaInt(16,ad[index].block);
        }
        
    }
}

__global__ void OCB128DecryptRandomAcces(aesBlock *m,aesBlock *delta, unsigned long long mlen,unsigned long long mlen2, unsigned long long deltalen, unsigned int *keys){
    int index = blockDim.x*blockIdx.x + threadIdx.x;
    unsigned long long deltaIndex = floor( (double) index/64);//esto se cambia para aes 1 ronda

    if( index<mlen2/16){
        __syncthreads();
        unsigned int deltaBlock[4];
        if(mlen%16==0){
            for (int i = 0 ; i< 4 ; i++){
                deltaBlock[i]= delta[deltaIndex].block[i]+index;
            }
            // falta añadir el checksum y si los bloques no vienen completos
    
            OCBAESDelta1Rounds(deltaBlock, keys);
    
            XOR_128(m[index].block,deltaBlock);
    
            AES_128Decrypt(m, keys,index);
    
            XOR_128(m[index].block,deltaBlock);
    }
    else{
        //penultimo bloque porque movi tag a funcion a parte exclusivo para el decifrado
        if(index == ((mlen2-16)/16) ){

            aesBlock *aestemp;
            aestemp = new aesBlock [1];
            for (int i = 0 ; i< 4 ; i++){
                aestemp[0].block[i] = delta[deltaIndex].block[i]+index+1;
            }
            
            OCBAESDelta1Rounds(aestemp[0].block, keys);


            AES_128(aestemp, keys,0);

            // imprimiArregloCudaInt(4,m[index].block);
            // imprimiArregloCudaInt(4,delta[deltaIndex].block);

            XOR_128(m[index].block,aestemp[0].block);
            
            for (int i = 0 ; i< 4 ; i++){
                aestemp[0].block[i] = 0;
            }
            memcpy(aestemp[0].block, m[index].block, mlen);
            memcpy(m[index].block, aestemp[0].block, 16);


        }
        else{
            for (int i = 0 ; i< 4 ; i++){
                deltaBlock[i]= delta[deltaIndex].block[i]+index;
            }
            // falta añadir el checksum y si los bloques no vienen completos
    
            OCBAESDelta1Rounds(deltaBlock, keys);
    
            XOR_128(m[index].block,deltaBlock);
    
            AES_128Decrypt(m, keys,index);
    
            XOR_128(m[index].block,deltaBlock);
        }
    }

    }
}

__global__ void OCB128DecryptRandomAccesGetTag(aesBlock *m,aesBlock *delta, aesBlock *S,bool mcomplete,unsigned long long mlen2, unsigned long long deltalen, unsigned int *keys){
    int index = blockDim.x*blockIdx.x + threadIdx.x;
    if( index==0){
        unsigned long long deltaIndex = floor( (double) ((mlen2/16) - 1)/64);//esto se cambia para aes 1 ronda
        __syncthreads();
        unsigned int deltaBlock[4];
        if(mcomplete){
            for (int i = 0 ; i< 4 ; i++){
                deltaBlock[i]= delta[deltaIndex].block[i]+((mlen2-16)/16)+2;
            }
    
            OCBAESDelta1Rounds(deltaBlock, keys);
    
            XOR_128(m[index].block,deltaBlock);

            AES_128(m, keys,index);
    
            XOR_128(m[index].block,S[0].block);
    }
    else{
        //ultimo bloque
            for (int i = 0 ; i< 4 ; i++){
                deltaBlock[i]= delta[deltaIndex].block[i]+ ((mlen2-16)/16) +3;
            }
            

            OCBAESDelta1Rounds(deltaBlock, keys);
    
            XOR_128(m[index].block,deltaBlock);

            AES_128(m, keys,index);
    
            XOR_128(m[index].block,S[0].block);
        
    }

    }
}

void OCBRandomAccess(aesBlock *m,aesBlock *delta, aesBlock *S, unsigned long long mlen,const unsigned long long mlen2, unsigned long long deltalen, unsigned int *keys){
    aesBlock *mCuda;
    aesBlock *SCuda;
    aesBlock *deltaCuda;
    unsigned int *keysCuda;

    int sizeMessage = (mlen2/16)*sizeof(class aesBlock);
    int sizeDelta = (deltalen)*sizeof(class aesBlock);
    int sizeS = sizeof(class aesBlock);
    int sizeKeys = 11*4*sizeof(unsigned int);

    hipMalloc(&mCuda, sizeMessage);
    hipMalloc(&keysCuda, sizeKeys);
    hipMalloc(&deltaCuda, sizeDelta);
    hipMalloc(&SCuda, sizeS);

    hipMemcpy(mCuda,m,sizeMessage,hipMemcpyDefault);
    hipMemcpy(deltaCuda,delta,sizeDelta,hipMemcpyDefault);
    hipMemcpy(keysCuda,keys,sizeKeys,hipMemcpyDefault);
    hipMemcpy(SCuda,S,sizeS,hipMemcpyDefault);

    dim3 nb( (unsigned int) ceil((double) mlen2/16.0) ) ; 
    dim3 nt(2); 
     

    OCB128EncryptRandomAcces<<<nb, nt>>>(mCuda,deltaCuda,SCuda, mlen, mlen2, deltalen,keysCuda);
    hipMemcpy(m, mCuda, sizeMessage, hipMemcpyDefault);

    hipFree(mCuda); 
    hipFree(keysCuda);  
    hipFree(deltaCuda);
    hipFree(SCuda);
}

void OCBRandomAccessAsociatedData(aesBlock *ad, aesBlock *delta, aesBlock *Ek1, const unsigned long long adlen, unsigned long long adlenMultiplo16,  unsigned long long deltalen, unsigned int *keys){
    aesBlock *adCuda;
    aesBlock *deltaCuda;
    aesBlock *Ek1Cuda;
    unsigned int *keysCuda;

    int sizeAd = (adlenMultiplo16/16)*sizeof(class aesBlock);
    int sizeDelta = (deltalen)*sizeof(class aesBlock);
    int sizeEk1 = sizeof(class aesBlock); //el result y el ek son del mismo tamaño
    int sizeKeys = 11*4*sizeof(unsigned int);

    hipMalloc(&adCuda, sizeAd);
    hipMalloc(&Ek1Cuda, sizeEk1);
    hipMalloc(&keysCuda, sizeKeys);
    hipMalloc(&deltaCuda, sizeDelta);

    hipMemcpy(adCuda,ad,sizeAd,hipMemcpyDefault);
    hipMemcpy(Ek1Cuda,Ek1,sizeEk1,hipMemcpyDefault);
    hipMemcpy(keysCuda,keys,sizeKeys,hipMemcpyDefault);
    hipMemcpy(deltaCuda,delta,sizeDelta,hipMemcpyDefault);

    dim3 nb( (unsigned int) ceil((double) adlenMultiplo16/16.0) ) ; 
    dim3 nt(2); 
     

    OCB128EncryptRandomAccesAsociatedData<<<nb, nt>>>(adCuda, deltaCuda, Ek1Cuda, adlen, adlenMultiplo16, deltalen,keysCuda);
    
    hipMemcpy(ad, adCuda, sizeAd, hipMemcpyDefault);

    hipFree(adCuda); 
    hipFree(keysCuda);  
    hipFree(deltaCuda); 
    hipFree(Ek1Cuda); 
}



void OCBRandomAccessDecrypt(aesBlock *m,aesBlock *delta, const unsigned long long clen,const unsigned long long mlen2, unsigned long long deltalen, unsigned int *keys){
    aesBlock *mCuda;
    aesBlock *deltaCuda;
    unsigned int *keysCuda;

    int sizeMessage = (mlen2/16)*sizeof(class aesBlock);
    int sizeDelta = (deltalen)*sizeof(class aesBlock);
    int sizeKeys = 11*4*sizeof(unsigned int);

    hipMalloc(&mCuda, sizeMessage);
    hipMalloc(&keysCuda, sizeKeys);
    hipMalloc(&deltaCuda, sizeDelta);

    hipMemcpy(mCuda,m,sizeMessage,hipMemcpyDefault);
    hipMemcpy(deltaCuda,delta,sizeDelta,hipMemcpyDefault);
    hipMemcpy(keysCuda,keys,sizeKeys,hipMemcpyDefault);

    dim3 nb( (unsigned int) ceil((double) mlen2/16.0) ) ; 
    dim3 nt(2); 
     

    OCB128DecryptRandomAcces<<<nb, nt>>>(mCuda,deltaCuda, clen, mlen2, deltalen,keysCuda);
    hipMemcpy(m, mCuda, sizeMessage, hipMemcpyDefault);

    hipFree(mCuda); 
    hipFree(keysCuda);  
    hipFree(deltaCuda);
}

void OCBRandomAccessGetTag(aesBlock *m,aesBlock *delta, aesBlock *S, bool mcomplete,const unsigned long long mlen2, unsigned long long deltalen, unsigned int *keys){
    aesBlock *mCuda;
    aesBlock *SCuda;
    aesBlock *deltaCuda;
    unsigned int *keysCuda;

    int sizeMessage = sizeof(class aesBlock);
    int sizeDelta = (deltalen)*sizeof(class aesBlock);
    int sizeS = sizeof(class aesBlock);
    int sizeKeys = 11*4*sizeof(unsigned int);

    hipMalloc(&mCuda, sizeMessage);
    hipMalloc(&keysCuda, sizeKeys);
    hipMalloc(&deltaCuda, sizeDelta);
    hipMalloc(&SCuda, sizeS);

    hipMemcpy(mCuda,m,sizeMessage,hipMemcpyDefault);
    hipMemcpy(deltaCuda,delta,sizeDelta,hipMemcpyDefault);
    hipMemcpy(keysCuda,keys,sizeKeys,hipMemcpyDefault);
    hipMemcpy(SCuda,S,sizeS,hipMemcpyDefault);

    dim3 nb( 1 ) ; 
    dim3 nt(2); 
     

    OCB128DecryptRandomAccesGetTag<<<nb, nt>>>(mCuda,deltaCuda,SCuda, mcomplete, mlen2, deltalen,keysCuda);
    hipMemcpy(m, mCuda, sizeMessage, hipMemcpyDefault);
    hipFree(mCuda); 
    hipFree(keysCuda);  
    hipFree(deltaCuda);
    hipFree(SCuda);
}

void getDelta(const unsigned int nonce[4],const unsigned long long mlen, aesBlock* delta,unsigned int *keys,unsigned long long deltalen ){
    
    for(int i = 0; i<deltalen; i++){
        
        for (int j = 0; j<4;j++){
            if(j==3){
                delta[i].block[j]= nonce[j]+i;
            }
            else{
                delta[i].block[j]= nonce[j];
            }

        }
    }
    AES128Encrypt(delta, deltalen*16, keys);


}

void copyMessageToAESBlock(aesBlock* encrypt, int numBlocks,const unsigned int m2[]){
    for(int i = 0; i<numBlocks; i++){
        for (int j = 0; j<4;j++){
            encrypt[i].block[j]=  m2[(i*4)+j];
        }
    }
}

/*
const unsigned char *in arreglo de entrada

unsigned int *out arreglo de salida

unsigned long long len tamaño real del arreglo

unsigned long long mlen2 tamaño en multiplo de 16 mas cercano al real
*/

void unsignedCharArrayTounsignedIntArray(const unsigned char *in,unsigned int *out, unsigned long long len, unsigned long long mlen2 ){
    
    unsigned char h[mlen2];
    unsigned char temp[mlen2];
	
    memcpy(h, in, mlen2);
    memcpy(temp, in, mlen2);
    
    int shifttab[16]= {
        12, 8, 4, 0,   
        13, 9, 5, 1,  
        14, 10, 6, 2,
        15, 11, 7, 3 
        };

    for(int i = 0; i < mlen2; i++){
        if(i<len ){
            int index = shifttab[i%16]+(floor(i/16)*16 );
            
            temp[i] = h[index];
            // printf("%x  \n",h[i]);

        } 
        else{
            temp[i]=0x00;
        }

    }
    
    unsigned int * temp2;
    temp2 = (unsigned int *) temp;
    for(int i = 0; i < mlen2/4; i++){
        out[i]=temp2[i];
    }
}



void checksum (aesBlock *in, unsigned long long tam, unsigned int *out ){
    
    for (int i=0; i<tam;i++){
        out[0] = out[0] ^ in[i].block[0];
        out[1] = out[1] ^ in[i].block[1];
        out[2] = out[2] ^ in[i].block[2];
        out[3] = out[3] ^ in[i].block[3];
    }
}
int crypto_aead_encrypt(
	unsigned char *c, unsigned long long *clen,
	const unsigned char *m, unsigned long long mlen,
	const unsigned char *ad, unsigned long long adlen,
	const unsigned char *nsec,
	const unsigned char *npub,
	const unsigned char *k){
    
    //calculo de bloques completos en tamaños de 16, siempre se completa a multiplo de 16
    
    int mlen2 = mlen;  
    if(mlen%16!=0)
        mlen2 = 16-(mlen%16)+ (mlen);
    int adlenMultiplo16 = adlen;  
    if(adlen%16!=0)
        adlenMultiplo16 = 16-(adlen%16)+ (adlen);

    unsigned long long bloques = (unsigned long long) ceil( (double) mlen2/16.0); //cada 4080 salta en 1 el delta
    unsigned long long deltalen = ceil( (double) mlen2/64.0);//el divisor cambia para 1 ronda de aes
   
    bool mcomplete = !(mlen%16);
    aesBlock* delta;
    delta = new aesBlock [deltalen];
    
    int numBlocks = mlen2/16;
    aesBlock* encrypt;
    encrypt = new aesBlock [numBlocks+1];//Se le suma uno por el bloque de la sumatoria

    aesBlock* Ek1;
    Ek1 = new aesBlock [1];

    aesBlock* S;
    S = new aesBlock [1];

    int numAdBlocks = adlenMultiplo16/16;
    aesBlock* asociateData;
    asociateData = new aesBlock [numAdBlocks];

    unsigned int nonce[4] = {0};


    unsigned int message[mlen2/4];
    unsigned int adTemp[adlenMultiplo16/4];
    unsigned int keys[11][4];
    unsigned int key[4];

    unsignedCharArrayTounsignedIntArray(k,key,16,16);
    unsignedCharArrayTounsignedIntArray(nsec,nonce,16,16);
    unsignedCharArrayTounsignedIntArray(m,message,mlen,mlen2);
    unsignedCharArrayTounsignedIntArray(ad,adTemp,adlen,adlenMultiplo16);
    

    for(int i = 0; i<numBlocks; i++){
        for (int j = 0; j<4;j++){
            encrypt[i].block[j]=  message[(i*4)+j];
        }
    }
    for(int i = 0; i<numAdBlocks; i++){
        for (int j = 0; j<4;j++){
            asociateData[i].block[j]=  adTemp[(i*4)+j];
        }
    }
    Ek1[0].block[0] = 0Xffffffff;
    Ek1[0].block[1] = 0Xffffffff;
    Ek1[0].block[2] = 0Xffffffff;
    Ek1[0].block[3] = 0Xffffffff;


    //expansion de llaves
    ExpansionKeys128(key,1, keys);
    //obetencion de la delta por medio del nonce 
    getDelta(nonce , mlen2, delta, &keys[0][0],deltalen);
    
    //calculo de Ek1 para lamda 5
    AES128Encrypt(Ek1, 16, &keys[0][0]);

    OCBRandomAccessAsociatedData(asociateData, delta, Ek1, adlen, adlenMultiplo16, deltalen, &keys[0][0]);
    
    checksum (asociateData, numAdBlocks, S[0].block );

    checksum (encrypt, numBlocks, encrypt[numBlocks].block );

    OCBRandomAccess(encrypt, delta,S, mlen, mlen2+16, deltalen, &keys[0][0]);

    cout<<endl;

    cout<<"Key          ";
    imprimiArreglo(4,key);
    printf("\n---------------------------");
    cout<<endl;

    // cout<<"Nonce        ";
    // imprimiArreglo(4,nonce2);
    // printf("\n---------------------------");
    // cout<<endl;

    cout<<"Plaintext    ";
    imprimiArreglo(ceil((mlen%16)/4.0),message);
    printf("\n---------------------------");
    cout<<endl;

    cout<<"Ciphertext   ";
    for(int i = 0; i<numBlocks; i++){
        if(i==numBlocks-1)
            imprimiArreglo( ceil((mlen%16)/4.0),encrypt[i].block);
        else
            imprimiArreglo(4,encrypt[i].block);
    }
    printf("\n---------------------------");
    cout<<endl;

    cout<<"Tag          ";
    imprimiArreglo(4,encrypt[numBlocks].block);
    cout<<endl;
    return 1;
}


int crypto_aead_decrypt(
	unsigned char *m, unsigned long long *mlen,
	unsigned char *nsec,
	const unsigned char *c, unsigned long long clen,
	const unsigned char *ad, unsigned long long adlen,
	const unsigned char *npub,
	const unsigned char *k){

    //calculo de bloques completos en tamaños de 16, siempre se completa a multiplo de 16
    int clen2 = clen;  
    if(clen2%16!=0)
        clen2 = 16-(clen%16)+ (clen);
    int adlenMultiplo16 = adlen;  
    if(adlen%16!=0)
        adlenMultiplo16 = 16-(adlen%16)+ (adlen);

    unsigned long long bloques = (unsigned long long) ceil( (double) clen2/16.0); //cada 4080 salta en 1 el delta
    unsigned long long deltalen = 1;//el divisor cambia para 1 ronda de aes
    //en esta version no hay problema porque es para dos rondas de aes con una ronda se tiene que cambiar
   
    bool mcomplete = !(clen%16);
    aesBlock* delta;
    delta = new aesBlock [deltalen];
  
    int numBlocks = clen2/16;
    aesBlock* encrypt;
    encrypt = new aesBlock [numBlocks];//Se le suma uno por el bloque de la sumatoria

    aesBlock* sumcheck;
    sumcheck = new aesBlock [1];//Se le suma uno por el bloque de la sumatoria

    aesBlock* Ek1;
    Ek1 = new aesBlock [1];

    aesBlock* S;
    S = new aesBlock [1];

    int numAdBlocks = adlen/16;
    aesBlock* asociateData;
    asociateData = new aesBlock [numAdBlocks];

    unsigned int nonce[4] = {0};


    unsigned int message[clen2/4];
    unsigned int adTemp[adlenMultiplo16/4];
    unsigned int keys[11][4];
    unsigned int key[4];

    unsignedCharArrayTounsignedIntArray(k,key,16,16);
    unsignedCharArrayTounsignedIntArray(c,message,clen,clen2);
    unsignedCharArrayTounsignedIntArray(nsec,nonce,16,16);
    unsignedCharArrayTounsignedIntArray(ad,adTemp,adlen,adlenMultiplo16);
    

    for(int i = 0; i<numBlocks; i++){
        for (int j = 0; j<4;j++){
            encrypt[i].block[j]=  message[(i*4)+j];
        }
    }
    for(int i = 0; i<numAdBlocks; i++){
        for (int j = 0; j<4;j++){
            asociateData[i].block[j]=  adTemp[(i*4)+j];
        }
        
    }

    Ek1[0].block[0] = 0Xffffffff;
    Ek1[0].block[1] = 0Xffffffff;
    Ek1[0].block[2] = 0Xffffffff;
    Ek1[0].block[3] = 0Xffffffff;


    //expansion de llaves
    ExpansionKeys128(key,1, keys);
    //obetencion de la delta por medio del nonce 
    getDelta(nonce , clen2, delta, &keys[0][0],deltalen);

    
    //calculo de Ek1 para lamda 5
    AES128Encrypt(Ek1, 16, &keys[0][0]);

    OCBRandomAccessAsociatedData(asociateData, delta, Ek1, adlen, adlenMultiplo16, deltalen, &keys[0][0]);
    
    checksum (asociateData, numAdBlocks, S[0].block );    

    OCBRandomAccessDecrypt(encrypt, delta, clen, clen2, deltalen, &keys[0][0]);

    checksum (encrypt, numBlocks, sumcheck[0].block );
    
    OCBRandomAccessGetTag(sumcheck, delta, S, mcomplete, clen2+16, deltalen, &keys[0][0]);

    cout<<endl;

    cout<<"Key          ";
    imprimiArreglo(4,key);
    printf("\n---------------------------");
    cout<<endl;

    cout<<"Ciphertext   ";
    imprimiArreglo(ceil((clen%16)/4.0),message);
    printf("\n---------------------------");
    cout<<endl;

    cout<<"Plaintext    ";
    for(int i = 0; i<numBlocks; i++){
        if(i==numBlocks-1)
        imprimiArreglo( ceil((clen%16)/4.0),encrypt[i].block);
        else
        imprimiArreglo(4,encrypt[i].block);
    }
    printf("\n---------------------------");
    cout<<endl;
    cout<<"Tag          ";
    imprimiArreglo(4,sumcheck[0].block);
    cout<<endl;
    

    return 1;
}

int main(int argc, char **argv) {
    

    const unsigned char k[16] ={ 
        0x2b,0x28,0xab,0x09,
        0x7e,0xae,0xf7,0xcf,
        0x15,0xd2,0x15,0x4f,
        0x16,0xa6,0x88,0x3c
    };
    const unsigned char m[16] ={ 
        0x30, 0x88, 0x6c, 0x7f,   
        0x32, 0x7f, 0xfe, 0xad, 
        0xee, 0xdf, 0x75, 0x48,
        0x6f, 0x09, 0xe7, 0xb6,
        // 0x32,0x88,0x31,0xe0,
        // 0x43,0x5a,0x31,0x37,
        // 0xf6,0x30,0x98,0x07,
        // 0xa8,0x8d,0xa2,0x34,
    };

    const unsigned char m2[32] ={ 
        0x32,0x88,0x31,0xe0,
        0x43,0x5a,0x31,0x37,
        0xf6,0x30,0x98,0x07,
        0xa8,0x8d,0xa2,0x34,

        0x32,0x88,0x31,0xe0,
        0x43,0x5a,0x31,0x37,
        0xf6,0x30,0x98,0x07,
        0xa8,0x8d,0xa2,0x34,
    };
    unsigned long long mlen=3;
   
    unsigned char c[32]={
        0x00, 0x00, 0x00, 0x00,   
        0xbb, 0x00, 0x00, 0x00, 
        0x61, 0x00, 0x00, 0x00,
        0x13, 0x00, 0x00, 0x00,

        0x30, 0x5c, 0x39, 0x50, 
        0xe1, 0x95, 0x7 , 0xff,
        0xab, 0xdd, 0x58, 0x70, 
        0xc2, 0xe2, 0xb1, 0x38, 
    };
    unsigned long long * clen = 0;
     
    const unsigned char ad[32] ={ 
        0x2b,0x28,0xab,0x09,
        0x7e,0xae,0xf7,0xcf,
        0x15,0xd2,0x15,0x4f,
        0x16,0xa6,0x88,0x3c,

        0x2b,0x28,0xab,0x09,
        0x7e,0xae,0xf7,0xcf,
        0x15,0xd2,0x15,0x4f,
        0x16,0xa6,0x88,0x3c
    };
    unsigned long long adlen = 32;

    const unsigned char nsec[16] = {
        0x32, 0x43, 0xf6, 0xa8,
        0X88, 0X5a, 0X30, 0X8d,
        0x31, 0x31, 0x98, 0xa2,
        0xe0, 0x37, 0x07, 0x35
    };
    unsigned char nsec2[16] = {
        0x32, 0x43, 0xf6, 0xa8,
        0X88, 0X5a, 0X30, 0X8d,
        0x31, 0x31, 0x98, 0xa2,
        0xe0, 0x37, 0x07, 0x35
    };;
    const unsigned char npub[1]={0}; 
    cout<<"encrypt"<<endl;
    crypto_aead_encrypt(c, clen, m2, mlen, ad, adlen, nsec, npub, k);
    cout<<"Decrypt"<<endl;
    crypto_aead_decrypt(c, clen, nsec2, c, mlen, ad, adlen, npub, k);
    //compile comand -march=native;

    return 0;
}

